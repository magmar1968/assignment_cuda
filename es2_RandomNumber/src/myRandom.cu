#include "hip/hip_runtime.h"
#include "myRandom.h"
// #define DEBUG
namespace rnd
{
    double MyRandomImplementation::genUniform(const double min, const double max)
    {  
        uint n = genUniformInt();
        #ifdef DEBUG
        double m = n/(double)_m;
        double p = m* (max - min);
        std::cerr << "max - min      "  << (max - min) << std::endl;
        std::cerr << "rand int       "  << n << std::endl;
        std::cerr << "n/UINT_MAX     "  << m << std::endl;
        std::cerr << "UINT_MAX and m "  << UINT_MAX << "  " << _m << std::endl;
        std::cerr << "m* (max - min) "  << p << std::endl;
        std::cerr << "p + min        "  << p + min << std::endl;
        std::cerr << "check          "  << n/(double)_m * (max - min) + min << "\n\n";
        #endif
        
        return n/(double)_m * (max - min) + min;
    }

    double MyRandomImplementation::genGaussian(const double mean, const double dev_std)
    {
        if(_storedValue)
        {
            _storedValue = false;
            return _value;
        }
        else
        {
            //insert second way 
            double u = genUniform(), v = genUniform();
            double num = (sqrt(-2 * log( u) ) * cos( v * (2 * M_PI)));
            _value =  (sqrt(-2 * log( u) ) * sin( v * (2 * M_PI))); //check
            _value = _value*dev_std + mean;
            _storedValue  = true;
            //normalize the number for the required mean and dev_std 
            return  num * dev_std  + mean;   
        }
    }


    GenLinCongruential::GenLinCongruential(uint seed, uint a, uint b, uint m )
        :_current(seed),_a(a),_b(b),_m(m)
    {
        MyRandomImplementation::setM(_m);
    }

    uint GenLinCongruential::genUniformInt()
    {
        return _current = ( _a * _current + _b) % _m;
    }

    GenTausworth::GenTausworth(uint seed, uint type, uint m)
        :_current(seed), _m(m)
    {
        MyRandomImplementation::setM(_m);
        if(seed < 128)
        {
            std::cerr<< "ERROR: in __FUNCTION__             \n"
                       << "       seed must be grater than 128\n";
            _status = false;
        }

        // parameter settings

        switch (type)
        {
        case TAUSWORTH_1:
            _k1 = TAUS_1_K1;
            _k2 = TAUS_1_K2;
            _k3 = TAUS_1_K3;
            break;
        case TAUSWORTH_2:
            _k1 = TAUS_2_K1;
            _k2 = TAUS_2_K2;
            _k3 = TAUS_2_K3;
            break;
        case TAUSWORTH_3:
            _k1 = TAUS_3_K1;
            _k2 = TAUS_3_K2;
            _k3 = TAUS_3_K3;       
        default:
            std::cerr << "ERROR: wrong tausworth input please use one of\n"
                      << "       the avaible macro TAUSWORTH_(0-2)      \n";
            break;
        }

    }

    uint GenTausworth::genUniformInt()
    {
        uint b    = (((_current << _k1) ^ _current ) >> _k2);
        return _current  = (((_current & _m ) << _k3) ^ b);
    }

    bool GenTausworth::getStatus() const
    {
        return _status;
    }

    // ---------------------------------------------------------------------------------------------

    GenCombined::GenCombined(uint seed1, uint seed2, uint seed3, uint seed4, uint m)
        :_seed1(seed1), _seed2(seed2), _seed3(seed3), _seed4(seed4), _m(m)
    {
        MyRandomImplementation::setM(_m);
        genT1 = GenTausworth(_seed1, TAUSWORTH_1, _m);
        genT2 = GenTausworth(_seed2, TAUSWORTH_2, _m);
        genT3 = GenTausworth(_seed3, TAUSWORTH_3, _m);

        genL1 = GenLinCongruential(_seed4);

        if(!genT1.getStatus() and genT2.getStatus() and genT3.getStatus())
        {
            std::cerr << "ERROR: in __FUNCTION__";
            _status = false; 
        }
    }


    uint GenCombined::genUniformInt()
    {
        return genT1.genUniformInt()^genT2.genUniformInt()^
               genT3.genUniformInt()^genL1.genUniformInt();
    }







}