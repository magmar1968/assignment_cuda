#include "hip/hip_runtime.h"
#include <fstream>
#include <cstdlib>
#include <iostream>
#include <random>
#include <string>
#include "lib.h"

struct Device
{
    bool GPU = 0;
    bool CPU = 0;
};



__global__ void gpuArraySum(float* a, float* b, float* c,int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    //check for overflow
    if (tid < n){
        c[tid] = a[tid] + b[tid];
    }
}

void cpuArraySum(float *a, float * b, float*c, int n)
{
    for( int i = 0; i < n; ++i)
    {
        c[i] = a[i] + b[i];
    }
}


int main(int argc, char ** argv)
{
    typedef const int cint;

    Device dev;
    // parse input
    if( cmdOptionExists(argv, argv + argc, "-h")){
        std::cout << "usage: -[option] [attibute]       \n" 
                  << "options: -h help                  \n" 
                  << "         -g select gpu as device  \n" 
                  << "         -c select cpu as device  \n" 
                  << "         -f change output filename\n";
    }
    if( cmdOptionExists(argv, argv + argc, "-cpu") or 
       !cmdOptionExists(argv, argv + argc, "-gpu")){
        dev.CPU = true;
    }
    if( cmdOptionExists(argv, argv + argc, "-gpu")){
        dev.GPU = true;
    }
    std::string filename;
    if( cmdOptionExists(argv, argv + argc, "-f"))
        filename = getCmdOption(argv, argv + argc, "-f");
    else
        filename = "timeseries.txt";

    std::fstream ofs(filename,std::fstream::out);
    
    // definition of the problem variable
    cint min_size   = 1000000;
    cint max_size   = 1000100;
    cint block_size = 256;
    cint iteration  = 10;
   
    //intialize the random engine
    std::random_device rnd;  
    std::default_random_engine eng(rnd());

    float *devA, *devB, *devC;

    std::cerr<< "fino a qui\n";
    int cont = 0;
    for (int N = min_size; N < max_size; ++N)
    {
        double time = 0;
        float A[N] = {};
        float B[N] = {};
        float C[N] = {};
        //create the space inside the  GPU memory
        if(dev.GPU)
        {
            hipMalloc( (void**)&devA, N*sizeof(float) );
            hipMalloc( (void**)&devB, N*sizeof(float) );
            hipMalloc( (void**)&devC, N*sizeof(float) );
        }
        
        cint grid_size = ((N + block_size)/block_size);
        for(int it = 0; it < iteration ; ++it)
        {
            ++ cont;
            std::cerr << "iterazione " <<  cont << std::endl;
            //gen vectors
            fillArray(A,N,eng);
            fillArray(B,N,eng);

            //start time    
            Timer myTimer;

            if(dev.GPU){
                hipMemcpy( devA, A, N*sizeof(float), hipMemcpyHostToDevice);
                hipMemcpy( devB, B, N*sizeof(float), hipMemcpyHostToDevice);
                gpuArraySum<<<grid_size,block_size>>>(devA, devB, devC,N);
                hipMemcpy(C, devC, N*sizeof(float), hipMemcpyDeviceToHost);
            }
            else
                cpuArraySum(A,B,C,N);

            //stop time
            time += myTimer.getTimeDiff();
        }
        //normalize to the number of iteration
        time/= iteration;
        // print on file
        ofs << N << "," << time << "\n";
        
        // free all the occupied memory
        if(dev.GPU)
            hipFree(devA);hipFree(devB);hipFree(devC);
    }


    return 0;
}




