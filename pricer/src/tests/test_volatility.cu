#include "hip/hip_runtime.h"
#include "header.cuh"



__host__             bool  run_device( prcr::Pricer_args *,prcr::Vol_args *);

__global__           void kernel(prcr::Pricer_args *,prcr::Vol_args *);

__host__             bool  simulate_host(prcr::Pricer_args *,prcr::Vol_args *);
__device__           void simulate_device(prcr::Pricer_args *, prcr::Volatility_surface *,prcr::Vol_args *);
__host__ __device__  void simulate_generic(size_t index, prcr::Volatility_surface *,prcr::Vol_args *);


__host__ bool
run_device(prcr::Pricer_args * prcr_args, prcr::Vol_args * host_vol_args)
{
    using namespace prcr;
    hipError_t cudaStatus;
    Pricer_args * dev_prcr_args;
    Vol_args    * dev_vol_args;

    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB    = prcr_args->dev_opts.N_threads;

    cudaStatus = hipMalloc((void**)&dev_prcr_args,sizeof(dev_prcr_args));
    if(cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc1 failed!\n");}

    cudaStatus = hipMalloc((void**)&dev_vol_args, NBLOCKS * TPB * sizeof(Vol_args));
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc2 failed!\n"); }
    
    
    cudaStatus = hipMemcpy(dev_prcr_args,prcr_args, sizeof(prcr_args),hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy1 failed!\n"); }
    
    cudaStatus = hipMemcpy(dev_vol_args,host_vol_args,  NBLOCKS*TPB*sizeof(Vol_args),hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy2 failed!\n"); }
    

    
    kernel <<< NBLOCKS, TPB>>>(dev_prcr_args,dev_vol_args);

    cudaStatus = hipMemcpy(host_vol_args, dev_vol_args, NBLOCKS*TPB*sizeof(Vol_args), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy3 failed!\n"); }

    bool kernel_error_check = true;
    for (int i = 0; i < NBLOCKS*TPB; i++)
    {
	    kernel_error_check = kernel_error_check && (host_vol_args[i].vol == prcr_args->vol_args.vol);  //controlla che simulate_generic abbia 										  //agito correttamente
        if(!kernel_error_check)
        {
            std::cerr << "something is going wrong\n";
        }
    }

    hipFree(dev_prcr_args);
    hipFree(dev_vol_args);

    return ((cudaStatus==hipSuccess) && (kernel_error_check));
}

__global__ void 
kernel(prcr::Pricer_args * prcr_args, prcr::Vol_args * vol_args)
{
    using namespace prcr;

    Volatility_surface * vol_srfc = new Volatility_surface(prcr_args->vol_args.vol);
    simulate_device(prcr_args,vol_srfc,vol_args);

    delete(vol_srfc);
}

__device__ void
simulate_device(prcr::Pricer_args        * prcr_args, 
                prcr::Volatility_surface * vol_sfrc,
                prcr::Vol_args           * vol_args)
{
    size_t index   = blockIdx.x * blockDim.x + threadIdx.x;
    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB     = prcr_args->dev_opts.N_threads;
    if (index < NBLOCKS * TPB) simulate_generic( index, vol_sfrc,vol_args);
}

__host__ bool
simulate_host(prcr::Pricer_args * prcr_args,
              prcr::Vol_args    * vol_args)
{
    using namespace prcr;
    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB = prcr_args->dev_opts.N_threads;
    Volatility_surface * vol_srfc = new Volatility_surface(prcr_args->vol_args.vol);


    for (int index = 0; index < NBLOCKS * TPB; index++){
        simulate_generic(index,vol_srfc,vol_args);
    }
    bool stat = true;
    for (int j = 0; j < NBLOCKS * TPB; j++)
    {
        stat = stat && (vol_args[j].vol == prcr_args->vol_args.vol); //controllo che simulate generic abbia sovrascritto correttamente il campo
    }
    return stat;
}


__device__ __host__ void
simulate_generic(size_t index,
                 prcr::Volatility_surface * vol_sfrc,
                 prcr::Vol_args *           vol_args)
{
    vol_args[index].vol = vol_sfrc->Get_volatility();
}

int main(int argc, char ** argv)
{
    using namespace prcr;
    std::string filename = "./data/infile_MC_test2.txt";
    Pricer_args * prcr_args = new Pricer_args;
    ReadInputOption(filename,prcr_args);

    bool GPU = prcr_args->dev_opts.GPU;
    bool CPU = prcr_args->dev_opts.CPU;

    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB    = prcr_args->dev_opts.N_threads;

    Vol_args * host_vol_args = new Vol_args[NBLOCKS*TPB];

    for(size_t inc = 0; inc < NBLOCKS*TPB; inc ++)
    {
        host_vol_args[inc].vol = 0.;
    }
    bool status_gpu = true, status_cpu = true;
    
    if(GPU == true)
        for(int i = 0; i < N_TEST_SIM; ++i)
            status_gpu = status_gpu && run_device(prcr_args,host_vol_args);
    
    for(size_t inc = 0; inc < NBLOCKS*TPB; inc ++)
    {
        host_vol_args[inc].vol = 0.;
    }

    if(CPU == true)
        for(int i = 0; i < N_TEST_SIM; ++i)
            status_cpu = status_cpu && simulate_host(prcr_args,host_vol_args);

    delete(prcr_args);

    if ( (status_gpu && status_cpu) == true){
        std::cout << "No errors encountered" << std::endl;
        return 0;
    }
    else if(status_gpu == false && status_cpu == false)    {
        std::cerr << "ERROR: gpu and cpu simulations didn't work properly\n";
        return -1;
    }
    else if(status_gpu == false && status_cpu == true ){
        std::cerr << "ERROR: gpu simulation didn't work properly\n";
        return -2;
    }
    else{
        std::cerr << "ERROR: cpu simulation didn't work properly\n";
        return -3;
    }

        
}
