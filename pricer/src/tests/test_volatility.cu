#include "hip/hip_runtime.h"
#include "header.cuh"



__host__             bool  run_device( prcr::Pricer_args *,prcr::Vol_args *);

__global__           void kernel(prcr::Pricer_args *,prcr::Vol_args *);

__host__             bool  simulate_host(prcr::Pricer_args *,prcr::Vol_args *);
__device__           void simulate_device(prcr::Pricer_args *, prcr::Volatility_surface *,prcr::Vol_args *);
__host__ __device__  void simulate_generic(size_t index, prcr::Volatility_surface *,prcr::Vol_args *);


__host__ bool
run_device(prcr::Pricer_args * prcr_args, prcr::Vol_args * host_vol_args)
{
    using namespace prcr;
    hipError_t cudaStatus;
    Pricer_args * dev_prcr_args;
    Vol_args    * dev_vol_args;

    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB    = prcr_args->dev_opts.N_threads;

    cudaStatus = hipMalloc((void**)&dev_prcr_args,sizeof(dev_prcr_args));
    if(cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc1 failed!\n");}

    cudaStatus = hipMalloc((void**)&dev_vol_args, NBLOCKS * TPB * sizeof(Vol_args));
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc2 failed!\n"); }

    
    cudaStatus = hipMemcpy(dev_prcr_args,prcr_args, sizeof(prcr_args),hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy1 failed!\n"); }
    fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));

    cudaStatus = hipMemcpy(dev_vol_args,host_vol_args,  NBLOCKS*TPB*sizeof(Vol_args),hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy2 failed!\n"); }
    fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));

    
    kernel <<< NBLOCKS, TPB>>>(dev_prcr_args,dev_vol_args);

    cudaStatus = hipMemcpy(host_vol_args, dev_vol_args, NBLOCKS*TPB*sizeof(Vol_args), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy3 failed!\n"); }
    fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));

    bool kernel_error_check = true;
    for (int i = 0; i < NBLOCKS*TPB; i++)
    {
	    kernel_error_check = kernel_error_check && (host_vol_args[i].vol == prcr_args->vol_args.vol);  //controlla che simulate_generic abbia 										  //agito correttamente
    }

    hipFree(dev_prcr_args);
    hipFree(dev_vol_args);

    return ((cudaStatus==hipSuccess) && (kernel_error_check));
}

__global__ void 
kernel(prcr::Pricer_args * prcr_args, prcr::Vol_args * vol_args)
{
    using namespace prcr;

    Volatility_surface * vol_srfc = new Volatility_surface(prcr_args->vol_args.vol);
    simulate_device(prcr_args,vol_srfc,vol_args);

    delete(vol_srfc);
}

__device__ void
simulate_device(prcr::Pricer_args        * prcr_args, 
                prcr::Volatility_surface * vol_sfrc,
                prcr::Vol_args           * vol_args)
{
    size_t index   = blockIdx.x * blockDim.x + threadIdx.x;
    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB     = prcr_args->dev_opts.N_threads;
    if (index < NBLOCKS * TPB) simulate_generic( index, vol_sfrc,vol_args);
}

__host__ bool
simulate_host(prcr::Pricer_args * prcr_args,
              prcr::Vol_args    * vol_args)
{
    using namespace prcr;
    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB = prcr_args->dev_opts.N_threads;
    Volatility_surface * vol_srfc = new Volatility_surface(prcr_args->vol_args.vol);


    for (int index = 0; index < NBLOCKS * TPB; index++){
        simulate_generic(index,vol_srfc,vol_args);
    }
    bool stat = true;
    for (int j = 0; j < NBLOCKS * TPB; j++)
    {
        stat = stat && (vol_args[j].vol == prcr_args->vol_args.vol); //controllo che simulate generic abbia sovrascritto correttamente il campo
    }
    return stat;
}


__device__ __host__ void
simulate_generic(size_t index,
                 prcr::Volatility_surface * vol_sfrc,
                 prcr::Vol_args *           vol_args)
{
    vol_args[index].vol = vol_sfrc->Get_volatility();
}

int main(int argc, char ** argv)
{
    using namespace prcr;
    std::string filename = "./data/infile_MC_test2.txt";
    Pricer_args * prcr_args = new Pricer_args;
    ReadInputOption(filename,prcr_args);

    bool GPU = prcr_args->dev_opts.GPU;
    bool CPU = prcr_args->dev_opts.CPU;
    bool status = true;

    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB    = prcr_args->dev_opts.N_threads;

    Vol_args * host_vol_args = new Vol_args[NBLOCKS*TPB];

    for(size_t inc = 0; inc < NBLOCKS*TPB; inc ++)
    {
        host_vol_args[inc].vol = 0.;
    }

    if(GPU == true)
        for(int i = 0; i < N_TEST_SIM; ++i)
            status = status && run_device(prcr_args,host_vol_args);
    
    if(CPU == true)
        for(int i = 0; i < N_TEST_SIM; ++i)
            status = status && simulate_host(prcr_args,host_vol_args);
    
        
    delete(prcr_args);
    return status;
}
