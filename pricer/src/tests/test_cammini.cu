#include "hip/hip_runtime.h"
#include "header.cuh"
#include <iomanip>

#define NSIM 1

bool __host__ run_device        (prcr::Pricer_args* prcr_args, double* host_last_steps);
void __global__ kernel          (prcr::Pricer_args* prcr_args, double* dev_last_steps);
bool __host__   simulate_host   (prcr::Pricer_args* prcr_args, double* host_last_steps);
void __device__ simulate_device (prcr::Pricer_args* prcr_args, prcr::Equity_prices*, prcr::Schedule*, double* dev_last_steps);
void __host__ __device__ simulate_generic
                                (size_t, prcr::Pricer_args* , prcr::Equity_prices*, prcr::Schedule*, double*);

__host__ bool
run_device(prcr::Pricer_args* prcr_args, double* host_last_steps)
{
    using namespace prcr;
    hipError_t cudaStatus;
    double* dev_last_steps;
    Pricer_args* dev_prcr_args;

    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB = prcr_args->dev_opts.N_threads;

    cudaStatus = hipMalloc((void**)&dev_prcr_args, sizeof(Pricer_args));
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc2 failed!\n"); }

    cudaStatus = hipMalloc((void**)&dev_last_steps, NBLOCKS * TPB * sizeof(double));
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc3 failed!\n"); }



    cudaStatus = hipMemcpy(dev_prcr_args, prcr_args, sizeof(Pricer_args), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy2 failed!\n"); }
    fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));
    cudaStatus = hipMemcpy(dev_last_steps, host_last_steps, NBLOCKS * TPB * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy3 failed!\n"); }
    fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));


    kernel << < NBLOCKS, TPB >> > (dev_prcr_args, dev_last_steps);

    cudaStatus = hipMemcpy(host_last_steps, dev_last_steps, NBLOCKS * TPB * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy4 failed!\n"); }
    fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));


    hipFree(dev_last_steps);
    hipFree(dev_prcr_args);

    return cudaStatus;
}




__global__ void
kernel(prcr::Pricer_args* prcr_args, double* dev_last_steps)
{
    using namespace prcr;

    Equity_description* descr = new Equity_description(
        prcr_args->eq_descr_args.dividend_yield,
        prcr_args->eq_descr_args.rate,
        prcr_args->eq_descr_args.vol);

    Equity_prices* starting_point = new Equity_prices(
        prcr_args->eq_price_args.time,
        prcr_args->eq_price_args.price,
        descr);

    Schedule* schedule = new Schedule(
        prcr_args->schedule_args.t_ref,
        prcr_args->schedule_args.deltat,
        prcr_args->schedule_args.dim);



    simulate_device( prcr_args, starting_point, schedule, dev_last_steps);

   
    delete(descr);
    delete(starting_point);
    delete(schedule);
}


__host__ bool
simulate_host(prcr::Pricer_args* prcr_args, double* last_steps)
{
    using namespace prcr;
    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB = prcr_args->dev_opts.N_threads;

    Equity_description* descr = new Equity_description(
        prcr_args->eq_descr_args.dividend_yield,
        prcr_args->eq_descr_args.rate,
        prcr_args->eq_descr_args.vol);

    Equity_prices* starting_point = new Equity_prices(
        prcr_args->eq_price_args.time,
        prcr_args->eq_price_args.price,
        descr);

    Schedule* schedule = new Schedule(
        prcr_args->schedule_args.t_ref,
        prcr_args->schedule_args.deltat,
        prcr_args->schedule_args.dim);


    for (int index = 0; index < NBLOCKS * TPB; ++index)
    {
        simulate_generic(index, prcr_args, starting_point, schedule, last_steps);
    }

    
    delete(descr);
    delete(starting_point);
    delete(schedule);
    return true; // da mettere gi� meglio
}


__device__ void
simulate_device(
    prcr::Pricer_args* prcr_args,
    prcr::Equity_prices* starting_point,
    prcr::Schedule* schedule,
    double* last_steps)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB = prcr_args->dev_opts.N_threads;
    if (index < NBLOCKS * TPB) simulate_generic(index, prcr_args, starting_point, schedule, last_steps);
}

__host__ __device__ void
simulate_generic(size_t index,
    prcr::Pricer_args* prcr_args,
    prcr::Equity_prices* starting_point,
    prcr::Schedule* schedule,
    double* last_steps)
{

    rnd::MyRandomDummy* gnr_in = new rnd::MyRandomDummy();
    prcr::Process_eq_lognormal* process 
                = new prcr::Process_eq_lognormal(gnr_in,prcr_args->stc_pr_args.exact);

                
    prcr::Path* path = new prcr::Path(starting_point,schedule,process);
    last_steps[index] = path->Get_last_eq_price();

    delete(path);
    delete(process);
    delete(gnr_in);
}




int main(int argc, char** argv)
{
    using namespace prcr;
    double exact_value = 100;   //capire da dove lo vogliamo ricavare

    srand(time(NULL));


    std::string filename = "./data/infile_test_cammini.txt";
    Pricer_args* prcr_args = new Pricer_args;
    ReadInputOption(filename, prcr_args);

    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB = prcr_args->dev_opts.N_threads;

    //last_steps
    double* last_steps = new double[NBLOCKS * TPB];   //array che contiene i valori del prezzo all'ultimo step, per ogni thread
    for (size_t inc = 0; inc < NBLOCKS * TPB; inc++)
    {
        last_steps[inc] = 0;
    }

    bool GPU = prcr_args->dev_opts.GPU;
    bool CPU = prcr_args->dev_opts.CPU;
    bool status = true;

    if (GPU == true) 
    {
        Timer gpu_timer;
        for(int i = 0; i < N_TEST_SIM; ++i)
            status = status && run_device(prcr_args, last_steps);
        gpu_timer.Stop();
    }

    if (CPU == true) 
    {
        Timer cpu_timer;
        for(int i = 0; i < N_TEST_SIM; ++i)
            status = status && simulate_host(prcr_args, last_steps);
        cpu_timer.Stop();
    }
    
    bool last_step_check = true;

    for (int j = 0; j < NBLOCKS * TPB; j++)
    {
        double delta = abs(last_steps[j] - exact_value);
        last_step_check = last_step_check && (delta < std::pow(10, -12));
    }
    std::cout << std::setprecision(12)<< "last step: " << last_steps[0] << "\n";
    delete[](last_steps);
    delete(prcr_args);
    
    return 0;    
}











