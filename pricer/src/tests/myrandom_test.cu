#include "hip/hip_runtime.h"
#include "../lib/support_lib/myRandom/myRandom_gnr/combined.cuh"
#include "../lib/support_lib/myRandom/myRandom.cuh"
//#include "../lib/support_lib/myRandom/random_numbers.cuh"
#include "../lib/support_lib/parse_lib/parse_lib.cuh"
#include <cmath>


//genero numeri casuali, li sommo e vedo se media � consistente
//genero numeri casuali a partire da seed noti e vedo se non cambiano

  

#define NBLOCKS 4
#define TPB 4
#define PPT 2

__global__ void kernel (uint*, double*, double*);
__device__ void rnd_test_dev(uint*, double*, double*);
__host__ void rnd_test_hst(uint*, double*, double*);
__host__ __device__ void rnd_test_generic(uint*, double*, double*, size_t);


__global__ void kernel(uint* seeds, double* dev_sum, double* dev_sq_sum)
{
    rnd_test_dev(seeds, dev_sum, dev_sq_sum);
}

__device__ void rnd_test_dev(uint* seeds, double* dev_sum, double* dev_sq_sum)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < NBLOCKS*TPB)
    {
        rnd_test_generic(seeds, dev_sum, dev_sq_sum, index);
    }
}
__host__ void rnd_test_hst(uint* seeds, double* sum, double* sq_sum)
{
    for(size_t index = 0; index < NBLOCKS*TPB; index++)
    rnd_test_generic(seeds, sum, sq_sum, index);
}
__host__ __device__ void rnd_test_generic(uint* seeds, double* sum, double* sq_sum, size_t index)
{
    uint seed0 = seeds[4 * index];
    uint seed1 = seeds[4 * index + 1];
    uint seed2 = seeds[4 * index + 2];
    uint seed3 = seeds[4 * index + 3];
    rnd::GenCombined* gnr = new rnd::GenCombined(seed0, seed1, seed2, seed3);
    double number;
    for (size_t i = 0; i < PPT; i++)
    {
        number = gnr->genGaussian();
        sum[index] += number;
        sq_sum[index] += number * number;
    }
    

}


int main(int argc, char** argv)
{

    prcr::Device dev;
    dev.CPU = false;
    dev.GPU = false;

    if (prcr::cmdOptionExists(argv, argv + argc, "-gpu"))
        dev.GPU = true;
    if (prcr::cmdOptionExists(argv, argv + argc, "-cpu"))
        dev.CPU = true;

    double host_sum[NBLOCKS * TPB];
    double host_sq_sum[NBLOCKS * TPB];
    uint seeds[4*NBLOCKS* TPB];

    srand(1);
    for (size_t i = 0; i < 4 * NBLOCKS * TPB; i++)
    {
        seeds[i] = rnd::genSeed(true);
    }
    for(size_t i = 0; i < NBLOCKS*TPB; i++)
    {
	host_sum[i] = 0;
	host_sq_sum[i] = 0;
    }




    if(dev.CPU)
    { 
        rnd_test_hst(seeds, host_sum, host_sq_sum);
    }



    if (dev.GPU)
    {
	    hipError_t cudaStatus;
        uint* dev_seeds = new uint[4*NBLOCKS*TPB];
        double* dev_sum = new double[NBLOCKS * TPB];
        double* dev_sq_sum = new double[NBLOCKS * TPB];


        cudaStatus = hipMalloc((void**)&dev_seeds, NBLOCKS *4* TPB * sizeof(uint));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc1 failed!\n"); }

        cudaStatus = hipMalloc((void**)&dev_sum,  NBLOCKS*TPB*sizeof(double));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc2 failed!\n"); }

        cudaStatus = hipMalloc((void**)&dev_sq_sum, NBLOCKS * TPB * sizeof(double));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc3 failed!\n"); }

        cudaStatus = hipMemcpy(dev_seeds, seeds, NBLOCKS * 4 *TPB* sizeof(uint), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy1 failed!\n"); }

        cudaStatus = hipMemcpy(dev_sum, host_sum, NBLOCKS  *TPB* sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy2 failed!\n"); }

        cudaStatus = hipMemcpy(dev_sq_sum, host_sq_sum, NBLOCKS *TPB* sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy3 failed!\n"); }

       kernel << <NBLOCKS,TPB >> > (dev_seeds, dev_sum, dev_sq_sum);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) { fprintf(stderr, "Kernel failed: %s\n", hipGetErrorString(cudaStatus)); }

        cudaStatus = hipMemcpy(host_sum, dev_sum, NBLOCKS*TPB*sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy back1 failed!\n"); }

        cudaStatus = hipMemcpy(host_sq_sum, dev_sq_sum, NBLOCKS * TPB * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy back2 failed!\n"); }


       
    }

    double std_dev = 1. / sqrt(NBLOCKS * TPB * PPT);
    std::cout << "std dev teorica (della media): " << std_dev << std::endl;

    double meas_mean = 0;
    double meas_std_dev = 0;

    for (size_t i = 0; i < NBLOCKS * TPB; i++)
    {
        meas_mean += host_sum[i];
        meas_std_dev += host_sq_sum[i];
    }

    meas_mean /= (NBLOCKS * TPB * PPT);
    meas_std_dev =sqrt((meas_std_dev/(NBLOCKS * TPB * PPT)-meas_mean*meas_mean)/(NBLOCKS*TPB*PPT));

    std::cout << " measured std dev:" << meas_std_dev<< std::endl;
    if (abs(meas_mean) < 3 * std_dev) 
    {
        printf("ok, ");
        printf("la media dei numeri generati e': %f\n", meas_mean);
        return 0;
    }
    else 
    {
        printf("La media non e' entro 3 standard deviation: %f\n", meas_mean);
        return 1;
    }
}



