#include "hip/hip_runtime.h"
#include "../lib/support_lib/myRandom/myRandom_gnr/combined.cuh"
#include "../lib/support_lib/myRandom/myRandom.cuh"
#include "../lib/support_lib/parse_lib/parse_lib.cuh"
#include "../lib/support_lib/timer_lib/myTimer.cuh"
#include <cmath>


//genero numeri casuali, li sommo e vedo se media � consistente
//genero numeri casuali a partire da seed noti e vedo se non cambiano

  

#define NBLOCKS 64
#define TPB 512
#define PPT 50

__global__ void kernel (uint*, double*, double*, bool*);
__device__ void rnd_test_dev(uint*, double*, double*, bool*);
__host__ void rnd_test_hst(uint*, double*, double*, bool*);
__host__ __device__ void rnd_test_generic(uint*, double*, double*, size_t, bool*);


__global__ void kernel(uint* seeds, double* dev_sum, double* dev_sq_sum, bool* cuda_bool)
{
    rnd_test_dev(seeds, dev_sum, dev_sq_sum, cuda_bool);
}

__device__ void rnd_test_dev(uint* seeds, double* dev_sum, double* dev_sq_sum, bool* cuda_bool)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < NBLOCKS*TPB)
    {
        rnd_test_generic(seeds, dev_sum, dev_sq_sum, index, cuda_bool);
	//__syncthreads();
    }
}
__host__ void rnd_test_hst(uint* seeds, double* sum, double* sq_sum, bool* host_bool)
{
    for(size_t index = 0; index < NBLOCKS*TPB; index++)
    rnd_test_generic(seeds, sum, sq_sum, index, host_bool);
}
__host__ __device__ void rnd_test_generic(uint* seeds, double* sum, double* sq_sum, size_t index, bool* status_bool)
{
    uint seed0 = seeds[4 * index];
    uint seed1 = seeds[4 * index + 1];
    uint seed2 = seeds[4 * index + 2];
    uint seed3 = seeds[4 * index + 3];

    rnd::GenCombined* gnr = new rnd::GenCombined(seed0, seed1, seed2, seed3);
    //rnd::GenCombined gnr(seed0, seed1, seed2, seed3);      
    //rnd::MyRandomDummy* gnr = new rnd::MyRandomDummy();   
    double number;
    for (size_t i = 0; i < PPT; i++)
    {
	if(gnr->Get_status() == false)
	*status_bool = false;
        else
        {
        	number = gnr->genGaussian();
		if((isnan(number))||(isinf(number)))
		{	
        		*status_bool = false;
		}	
		else
        	{
        		sum[index] += number;
        		sq_sum[index] += number*number;
       	    }
	
        }
    }
    delete(gnr);
}


int main(int argc, char** argv)
{
    prcr::Device dev;
    dev.CPU = false;
    dev.GPU = false;

    if (prcr::cmdOptionExists(argv, argv + argc, "-gpu"))
        dev.GPU = true;
    if (prcr::cmdOptionExists(argv, argv + argc, "-cpu"))
        dev.CPU = true;

    double* host_sum = new double[NBLOCKS * TPB];
    double* host_sq_sum =new double[NBLOCKS * TPB];
    uint* seeds = new uint [4*NBLOCKS* TPB];
        

    bool* host_cuda_bool = new bool;
    *host_cuda_bool = true;
    srand(1);  //CPU and GPU results must be the same (but not when srand(time(NULL)))
    uint seed_aus[4];
    for( size_t i = 0; i < 4; i++)
    {
    	seed_aus[i] = rnd::genSeed(true);
    }
    rnd::GenCombined gnr_aus(seed_aus[0], seed_aus[1], seed_aus[2], seed_aus[3]);
    for (size_t i = 0; i < 4 * NBLOCKS * TPB; i++)
    {
        seeds[i] = gnr_aus.genUniformInt();
	while(seeds[i] <=128)
	{
		seeds[i] = gnr_aus.genUniformInt();
        }
    }
    for(size_t i = 0; i < NBLOCKS*TPB; i++)
    {
	host_sum[i] = 0;
	host_sq_sum[i] = 0;
    }




    if(dev.CPU)
    { 
        Timer cpu_timer;
        rnd_test_hst(seeds, host_sum, host_sq_sum, host_cuda_bool);
        cpu_timer.Stop();
    }



    if (dev.GPU)
    {
	hipError_t cudaStatus;
        uint* dev_seeds = new uint[4*NBLOCKS*TPB];
        double* dev_sum = new double[NBLOCKS * TPB];
        double* dev_sq_sum = new double[NBLOCKS * TPB];
        bool* dev_cuda_bool = new bool;
        
        //cudaStatus = hipDeviceSetLimit(hipLimitMallocHeapSize, sizeof(??)); //?
        //if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaSetLimit failed\n"); }
   
        cudaStatus = hipMalloc((void**)&dev_seeds, NBLOCKS *4* TPB * sizeof(uint));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc1 failed!\n"); }

        cudaStatus = hipMalloc((void**)&dev_cuda_bool, sizeof(bool));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc2 failed!\n"); }

        cudaStatus = hipMalloc((void**)&dev_sum,  NBLOCKS*TPB*sizeof(double));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc3 failed!\n"); }

        cudaStatus = hipMalloc((void**)&dev_sq_sum, NBLOCKS * TPB * sizeof(double));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc4 failed!\n"); }

        cudaStatus = hipMemcpy(dev_seeds, seeds, NBLOCKS * 4 *TPB* sizeof(uint), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy5 failed!\n"); }

        cudaStatus = hipMemcpy(dev_sum, host_sum, NBLOCKS  *TPB* sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy6 failed!\n"); }

        cudaStatus = hipMemcpy(dev_sq_sum, host_sq_sum, NBLOCKS *TPB* sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy7 failed!\n"); }

        cudaStatus = hipMemcpy(dev_cuda_bool, host_cuda_bool, sizeof(bool), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy8 failed!\n"); }


        Timer gpu_timer;
        kernel << <NBLOCKS,TPB >> > (dev_seeds, dev_sum, dev_sq_sum, dev_cuda_bool);
        gpu_timer.Stop(); 
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) { fprintf(stderr, "Kernel failed: %s\n", hipGetErrorString(cudaStatus)); }


        cudaStatus = hipMemcpy(host_sum, dev_sum, NBLOCKS*TPB*sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy back1 failed! %s\n", hipGetErrorString(cudaStatus)); }
        cudaStatus = hipMemcpy(host_sq_sum, dev_sq_sum, NBLOCKS * TPB * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy back2 failed!\n"); }

        cudaStatus = hipMemcpy(host_cuda_bool, dev_cuda_bool, sizeof(bool), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy back3 failed!\n"); }

        hipFree(dev_seeds);
        hipFree(dev_sum);
        hipFree(dev_sq_sum);
        hipFree(dev_cuda_bool);
        /*delete[](dev_seeds);
        delete[](dev_sum);
        delete[](dev_sq_sum);
        delete(dev_cuda_bool);*/
    }

    if (!*host_cuda_bool)
    {
        printf("Something went wrong... (nan o inf generated)\n");
    }

    double std_dev = 1. / sqrt(NBLOCKS * TPB );
    std_dev /= sqrt(PPT);
    

    double meas_mean = 0;
    double meas_std_dev = 0;

    for (size_t i = 0; i < NBLOCKS * TPB; i++)
    {
	    //std::cout << host_sum[i] <<std::endl;
        meas_mean += host_sum[i];
        meas_std_dev += host_sq_sum[i];
    }
   
    meas_mean /= double (NBLOCKS * TPB)  ;
    meas_mean /= double(PPT);
    meas_std_dev /= double(NBLOCKS*TPB);
    meas_std_dev = sqrt((meas_std_dev/(PPT)-meas_mean*meas_mean)/double(NBLOCKS*TPB));
    meas_std_dev /= sqrt(PPT);


    delete[](seeds);
    delete[](host_sum);
    delete[](host_sq_sum);
    delete(host_cuda_bool);


    std::cout << "std dev teorica (della media): " << std_dev << std::endl;
    std::cout << " measured std dev:" << meas_std_dev<< std::endl;
    if (abs(meas_mean) < 3 * std_dev) 
    {
        printf("ok, ");
        printf("la media dei numeri generati e': %.3e\n", meas_mean);
        return 0;
    }
    else 
    {
        printf("La media non e' entro 3 standard deviation: %.3e\n", meas_mean);
        return 1;
    }

}



