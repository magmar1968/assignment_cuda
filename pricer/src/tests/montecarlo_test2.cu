#include "hip/hip_runtime.h"
#include <iostream>
#include ""
#include "../lib/path_gen_lib/path/path.cuh"
#include "../lib/support_lib/myRandom/myRandom.cuh"
#include "../lib/support_lib/myRandom/myRandom_gnr/combined.cuh"
#include "../lib/support_lib/myRandom/myRandom_gnr/tausworth.cuh"
#include "../lib/support_lib/myRandom/myRandom_gnr/linCongruential.cuh"
#include "../lib/path_gen_lib/process_eq_imp/process_eq_lognormal.cuh"
#include "../lib/equity_lib/schedule_lib/schedule.cuh"
#include "../lib/equity_lib/yield_curve_lib/yield_curve.cuh"
#include "../lib/equity_lib/yield_curve_lib/yield_curve_flat.cuh"
#include "../lib/equity_lib/yield_curve_lib/yield_curve_term_structure.cuh"
#include "../lib/support_lib/parse_lib/parse_lib.cuh"
#include "../lib/option_pricer_lib/option_pricer.cuh"
#include "../lib/option_pricer_lib/option_pricer_montecarlo/option_pricer_montecarlo.cuh"
#include "../lib/contract_option_lib/contract_eq_option_vanilla/contract_eq_option_vanilla.cuh"
#include "../lib/support_lib/statistic_lib/statistic_lib.cuh"
#include "../lib/support_lib/myDouble_lib/myudouble.cuh"
#include "../lib/support_lib/parse_lib/parse_lib.cuh"
#include "../lib/support_lib/timer_lib/myTimer.cuh"

struct Result
{
    double opt_price;
    double error;
};

__host__ bool run_device(uint * seeds, prcr::Pricer_args * prcr_args,Result * host_results);
void __global__ kernel(uint * seeds, prcr::Pricer_args * prcr_args,Result * dev_results);
bool __host__   simulate_host  (uint* seeds, prcr::Pricer_args* prcr_args, Result* dev_res);
void __device__ simulate_device(uint* seeds, prcr::Contract_eq_option_vanilla * contr_opt, 
                                prcr::Pricer_args * prcr_args, Result * dev_res); 
void __host__ __device__ 
                simulate_generic(uint * seeds, size_t index, 
                                 prcr::Contract_eq_option_vanilla * contr_opt,
                                 prcr::Pricer_args * prcr_args,
                                 Result *  results);

__host__ bool 
run_device(uint * seeds, prcr::Pricer_args * prcr_args,Result * host_res)
{   
    using namespace prcr;
    hipError_t cudaStatus;
    uint        * dev_seeds;
    Result      * dev_res;
    Pricer_args * dev_prcr_args;

    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB    = prcr_args->dev_opts.N_threads;

    cudaStatus = hipMalloc((void**)&dev_seeds, NBLOCKS * TPB * 4 * sizeof(uint));
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc1 failed!\n"); }

    cudaStatus = hipMalloc((void**)&dev_prcr_args,sizeof(dev_prcr_args));
    if(cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc2 failed!\n");  }

    cudaStatus = hipMalloc((void**)&dev_res, NBLOCKS * TPB * sizeof(Result));
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc3 failed!\n"); }



    cudaStatus = hipMemcpy(dev_seeds, seeds, NBLOCKS * TPB * 4 * sizeof(uint), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy1 failed!\n"); }
    fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));

    cudaStatus = hipMemcpy(dev_prcr_args,prcr_args, sizeof(prcr_args),hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy2 failed!\n"); }
    fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));

    cudaStatus = hipMemcpy(dev_res, host_res, NBLOCKS*TPB*sizeof(Result), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy3 failed!\n"); }
    fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));


    kernel <<< NBLOCKS, TPB>>>(dev_seeds,dev_prcr_args,dev_res);

    cudaStatus = hipMemcpy(host_res, dev_res, NBLOCKS*TPB*sizeof(Result), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy4 failed!\n"); }
    fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));


    hipFree(dev_seeds);
    hipFree(dev_res);
    hipFree(dev_prcr_args);

    return cudaStatus;
}




__global__ void 
kernel(uint * seeds, prcr::Pricer_args * prcr_args,Result * dev_results)
{
    using namespace prcr;

    Volatility_surface * volatility_surface = new Volatility_surface(prcr_args->vol_args.vol);

    Yield_curve_flat * yield_curve = new Yield_curve_flat(
                                          prcr_args->eq_descr_args.currency,
                                          prcr_args->yc_args.rate);

    Equity_description * descr = new Equity_description(
                                    prcr_args->eq_descr_args.isin_code,
                                    prcr_args->eq_descr_args.name,
                                    prcr_args->eq_descr_args.currency,
                                    prcr_args->eq_descr_args.dividend_yield,
                                    yield_curve,
                                    volatility_surface);
    
    Equity_prices * starting_point = new Equity_prices(
                                    prcr_args->eq_price_args.price,
                                    prcr_args->eq_price_args.price,
                                    descr);

    Schedule * schedule = new Schedule(
                                    prcr_args->schedule_args.t_ref,
                                    prcr_args->schedule_args.deltat,
                                    prcr_args->schedule_args.dim);
    
    Contract_eq_option_vanilla * eq_option = new Contract_eq_option_vanilla(
                                    starting_point,
                                    schedule,
                                    prcr_args->contract_args.strike_price,
                                    prcr_args->contract_args.contract_type);


    simulate_device(seeds,eq_option,prcr_args, dev_results);

    delete(volatility_surface);
    delete(yield_curve);
    delete(descr);
    delete(starting_point);
    delete(schedule);
    delete(eq_option);
}


__host__ bool 
simulate_host(uint* seeds, prcr::Pricer_args* prcr_args, Result* host_res)
{
    using namespace prcr;
    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB    = prcr_args->dev_opts.N_threads;

    Volatility_surface* volatility_surface = new Volatility_surface(prcr_args->vol_args.vol);

    Yield_curve_flat* yield_curve = new Yield_curve_flat(
        prcr_args->eq_descr_args.currency,
        prcr_args->yc_args.rate);

    Equity_description* descr = new Equity_description(
        prcr_args->eq_descr_args.isin_code,
        prcr_args->eq_descr_args.name,
        prcr_args->eq_descr_args.currency,
        prcr_args->eq_descr_args.dividend_yield,
        yield_curve,
        volatility_surface);

    Equity_prices* starting_point = new Equity_prices(
        prcr_args->eq_price_args.time,
        prcr_args->eq_price_args.price,
        descr);

    Schedule* schedule = new Schedule(
        prcr_args->schedule_args.t_ref,
        prcr_args->schedule_args.deltat,
        prcr_args->schedule_args.dim);

    Contract_eq_option_vanilla* contr_opt = new Contract_eq_option_vanilla(
        starting_point,
        schedule,
        prcr_args->contract_args.strike_price,
        prcr_args->contract_args.contract_type);

    for(int index = 0; index < NBLOCKS*TPB; ++index )
    {
        simulate_generic(seeds, index, contr_opt, prcr_args,host_res);

    }

    delete(volatility_surface);
    delete(yield_curve);
    delete(descr);
    delete(starting_point);
    delete(schedule);
    delete(contr_opt);
    return true; // da mettere giù meglio
}


__device__ void
simulate_device(uint * seeds, 
                prcr::Contract_eq_option_vanilla * contr_opt,
                prcr::Pricer_args * prcr_args,
                Result         * results)
{
    size_t index   = blockIdx.x * blockDim.x + threadIdx.x;
    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB     = prcr_args->dev_opts.N_threads;
    if (index < NBLOCKS * TPB) simulate_generic(seeds, index, contr_opt,prcr_args, results);    
}

__host__ __device__ void 
simulate_generic(uint * seeds, size_t index, 
                 prcr::Contract_eq_option_vanilla * contr_opt, 
                 prcr::Pricer_args * prcr_args,
                 Result * results)
{
    uint seed0 = seeds[0 + index * 4];
    uint seed1 = seeds[1 + index * 4];
    uint seed2 = seeds[2 + index * 4];
    uint seed3 = seeds[3 + index * 4];
    size_t PPT = prcr_args->mc_args.N_simulations;

    rnd::GenCombined               * gnr_in  = new rnd::GenCombined(seed0, seed1, seed2, seed3);
    prcr::Process_eq_lognormal     * process = new prcr::Process_eq_lognormal(gnr_in,false,1);
    prcr::Option_pricer_montecarlo * pric    = new prcr::Option_pricer_montecarlo(contr_opt,process,PPT);
    results[index].opt_price = pric->Get_price();
    results[index].error     = pric->Get_price_square();//MC_error();

    delete(gnr_in);
    delete(process);
    delete(pric);
}




int main(int argc, char ** argv)
{
    using namespace prcr;
    srand(time(NULL));
    
    
    std::string filename = "./data/infile_MC_test2.txt";
    Pricer_args * prcr_args = new Pricer_args;
    ReadInputOption(filename,prcr_args);
    
    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB    = prcr_args->dev_opts.N_threads;
    size_t PPT = prcr_args->mc_args.N_simulations;
    //seeeds generation
    uint* seeds = new uint[4 * NBLOCKS * TPB];
    for (size_t inc = 0; inc < 4 * NBLOCKS * TPB; inc++)
        seeds[inc] = rnd::genSeed(true);
    //results
    Result* host_res = new Result[NBLOCKS * TPB];
    for(size_t inc = 0; inc < NBLOCKS*TPB; inc ++)
    {
        host_res[inc].opt_price = 0;
        host_res[inc].error = 0;
    }

    bool GPU = prcr_args->dev_opts.GPU;
    bool CPU = prcr_args->dev_opts.CPU;
    bool status = true;

    if(GPU == true){ 
	Timer gpu_timer;
        status = status && run_device(seeds,prcr_args,host_res);
	gpu_timer.Stop();
    }
    
    if(CPU == true){
        status = status && simulate_host(seeds,prcr_args,host_res);

        double final_error = 0;
	    double squares_sum = 0;
        double final_price = 0;
        for( int i = 0 ; i < NBLOCKS* TPB; ++i)
        {
            //final_error += host_res[i].error;
            final_price += host_res[i].opt_price; 
    	    squares_sum += host_res[i].error;
	    
        }
        final_price /= static_cast<double>(NBLOCKS*TPB);
        //final_error /= static_cast<double>(NBLOCKS*TPB);
	    final_error = compute_final_error(squares_sum, final_price, NBLOCKS*TPB*PPT);
        std::cout << " CPU simulation final results:         \n"
                  << "         - price: " << final_price << "\n"
                  << "         - error: " << final_error << "\n";
    }

    delete[](host_res);
    delete[](seeds);
    delete(prcr_args);
    return status;
}











