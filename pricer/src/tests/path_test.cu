#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "../lib/path_gen_lib/path/path.cuh"
#include "../lib/support_lib/myRandom/myRandom.cuh"
#include "../lib/support_lib/myRandom/myRandom_gnr/combined.cuh"
#include "../lib/support_lib/myRandom/myRandom_gnr/tausworth.cuh"
#include "../lib/support_lib/myRandom/myRandom_gnr/linCongruential.cuh"
#include "../lib/path_gen_lib/process_eq_imp/process_eq_lognormal_multivariante.cuh"
#include "../lib/path_gen_lib/process_eq_imp/process_eq_lognormal.cuh"
#include "../lib/equity_lib/schedule_lib/schedule.cuh"
#include "../lib/equity_lib/yield_curve_lib/yield_curve.cuh"
#include "../lib/equity_lib/yield_curve_lib/yield_curve_flat.cuh"
#include "../lib/support_lib/parse_lib/parse_lib.cuh"


#define NPATH 5  //number of paths
#define STEPS 5  // number of steps
#define NEQ 5  //number of equities


__global__ void kernel(double*);
D void createPath_device(Process_eq*, Equity_prices*,  Schedule*, Path*, size_t, double*);
H void createPath_host(Process_eq*, Equity_prices*, Schedule*, Path*, size_t);
HD void createPath_generic(Process_eq*, Equity_prices*, Schedule*, Path* , size_t);




__global__ void kernel(double* path_out)
{
    rnd::GenCombined* gnr_in = new rnd::GenCombined(800, 200, 400, 500);
    Process_eq_lognormal_multivariante* process_in = new Process_eq_lognormal_multivariante(gnr_in, NEQ);
    pricer::udb start_prices[NEQ];
    for (int i = 0; i < NEQ; i++)
    {
        start_prices[i] = 100 * (1 + i) + i;
    }
    double start_time = 0.15;
    Equity_description** descr = new Equity_description * [NEQ];

    Volatility_surface* vol = new Volatility_surface(0.01);
    Yield_curve_flat* yc = new Yield_curve_flat("euro", 0);

    for (int i = 0; i < NEQ; i++)
    {
        descr[i] = new Equity_description;
        descr[i]->Set_isin_code("isin codein");
        descr[i]->Set_name("namein ");
        descr[i]->Set_currency("currencyin");
        descr[i]->Set_dividend_yield(0);
        descr[i]->Set_yc(yc);
        descr[i]->Set_vol_surface(vol);
    }

    Equity_prices* starting_point_in = new Equity_prices(start_time, start_prices, NEQ, descr);
    double tempi[STEPS];
    for (size_t k = 0; k < STEPS; k++)
    {
        tempi[k] = 0.2 + k * 0.2;
    }
    Schedule* calen = new Schedule(tempi, STEPS);

    Path* cammini_GPU = new Path[NPATH];

    createPath_device(process_in, starting_point_in, calen, cammini_GPU, NPATH, path_out);
}

D void createPath_device(Process_eq* process,
    Equity_prices* starting_point,
    Schedule* calendar,
    Path* cammini,
    size_t totpaths,
    double* path_out)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    while (index < totpaths)
    {
        createPath_generic(process, starting_point, calendar, cammini, index);
	
	for (size_t j = 0; j < NEQ; j++)
        {
            //path_out[i] 0 = new double[STEPS];
            for (size_t k = 0; k < STEPS; k++)
            {
                path_out[index] = cammini[index].Get_equity_prices(0)->Get_eq_price(0).get_number();
            }
        }
        index += blockDim.x * gridDim.x;
    }
}

H void createPath_host(Process_eq* process,
    Equity_prices* starting_point,
    Schedule* calendar,
    Path* cammini,
    size_t totpaths)
{
    for (size_t index = 0; index < totpaths; index++)
    {
        createPath_generic(process, starting_point, calendar, cammini, index);
    }
}


HD void createPath_generic(Process_eq* process,
    Equity_prices* starting_point,
    Schedule* calendar,
    Path* cammini,
    size_t index)
{
        cammini[index] = Path(starting_point, calendar, process);
}


int main(int argc, char **argv)
{

    srand(time(NULL));
    hipError_t cudaStatus;

    size_t* npath = new size_t(NPATH);
    size_t* neq = new size_t(NEQ);

    prcr::Device dev;
    dev.CPU = false;
    dev.GPU = false;

    if (prcr::cmdOptionExists(argv, argv + argc, "-gpu"))
        dev.GPU = true;
    if (prcr::cmdOptionExists(argv, argv + argc, "-cpu"))
        dev.CPU = true;

    if (dev.CPU == true)
    {

        rnd::GenCombined* gnr_in = new rnd::GenCombined(800, 200, 400, 500);
        Process_eq_lognormal_multivariante* process_in = new Process_eq_lognormal_multivariante(gnr_in, NEQ);
        pricer::udb start_prices[NEQ];
        for (int i = 0; i < NEQ; i++)
        {
            start_prices[i] = 100 * (1 + i) + i;
        }
        double start_time = 0.15;
        Equity_description** descr = new Equity_description * [NEQ];

        Volatility_surface* vol = new Volatility_surface(0.01);
        Yield_curve_flat* yc = new Yield_curve_flat("euro", 0);

        for (int i = 0; i < NEQ; i++)
        {
            descr[i] = new Equity_description;
            descr[i]->Set_isin_code("isin codein");
            descr[i]->Set_name("namein ");
            descr[i]->Set_currency("currencyin");
            descr[i]->Set_dividend_yield(0);
            descr[i]->Set_yc(yc);
            descr[i]->Set_vol_surface(vol);
        }

        Equity_prices* starting_point_in = new Equity_prices(start_time, start_prices, NEQ, descr);
        double tempi[STEPS];
        for (size_t k = 0; k < STEPS; k++)
        {
            tempi[k] = 0.2 + k * 0.2;
        }
        Schedule* calen = new Schedule(tempi, STEPS);




        Path* cammini_CPU = new Path[NPATH];
        createPath_host(process_in, starting_point_in, calen, cammini_CPU, NPATH);


        //stampa CPU

        std::cout << std::endl << "paths:" << std::endl;
        for (int i = 0; i < NPATH; i++)
        {
            std::cout << "\n\n\n\npath " << i << ":" << std::endl;
            for (int k = 0; k < NEQ; k++)
            {
                std::cout << "\nequity " << k << ":" << std::endl;
                for (int j = 0; j < STEPS; j++)
                    std::cout << cammini_CPU[i].Get_equity_prices(j)->Get_eq_price(k).get_number() << " ";
            }
            std::cout << std::endl;
            std::cout << std::endl;
        }
    }










    if (dev.GPU == true)
    {
        //CudaSetDevice(0);
        double* dev_paths;
        double* paths = new double[NPATH];
	/*for (size_t t = 0; t < NPATH; t++)
	{
		paths[t] = new double*[STEPS];
		for(size_t y = 0; y < STEPS; y++)
		{
			paths[t][y] = new double[NEQ];
		}
	}*/ 	

        cudaStatus = hipMalloc((void**)&dev_paths, NPATH*sizeof(double));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!\n"); }

        kernel << <32, 32 >> > (dev_paths);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) { fprintf(stderr, "Kernel failed: %s\n", hipGetErrorString(cudaStatus)); }

        cudaStatus = hipMemcpy(paths,dev_paths, NPATH * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!\n"); }
	fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));
        //stampa  GPU

        std::cout << std::endl << "paths:" << std::endl;
        for (int i = 0; i < NPATH; i++)
        {
            std::cout << "\n\n\n\npath " << i << ":" << std::endl;
            for (int k = 0; k < NEQ; k++)
            {
                std::cout << "\nequity " << k << ":" << std::endl;
                for (int j = 0; j < STEPS; j++)
                    std::cout << paths[i];
            }
            std::cout << std::endl;
            std::cout << std::endl;
        }

    }
    


    return 0;

}


/*cudaStatus = hipMalloc((void**)&dev_process, sizeof(Process_eq));
if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc 1 failed!\n"); }

cudaStatus = hipMalloc((void**)&dev_prices, sizeof(Equity_prices));
if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc 2 failed!\n"); }

cudaStatus = hipMalloc((void**)&dev_schedule, sizeof(Schedule));
if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc 3 failed!\n"); }

cudaStatus = hipMalloc((void**)&dev_totpaths, sizeof(size_t));
if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc 4 failed!\n"); }

cudaStatus = hipMalloc((void**)&dev_paths, NPATH * sizeof(Path));
if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc 6 failed!\n"); }




cudaStatus = hipMemcpy(dev_process, process_in, sizeof(Process_eq), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy 1 failed!\n"); }

cudaStatus = hipMemcpy(dev_prices, starting_point_in, sizeof(Equity_prices), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy 2 failed!\n"); }

cudaStatus = hipMemcpy(dev_schedule, calen, sizeof(Schedule), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy 3 failed!\n"); }

cudaStatus = hipMemcpy(dev_totpaths, npath, sizeof(size_t), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy 4 failed!\n"); }



kernel << <32, 32 >> > (process_in, starting_point_in, calen, cammini_GPU, NPATH);
cudaStatus = hipGetLastError();
if (cudaStatus != hipSuccess) { fprintf(stderr, "Kernel failed: %s\n", hipGetErrorString(cudaStatus)); }


cudaStatus = hipMemcpy(cammini_GPU, dev_paths, NPATH * sizeof(Path), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy backward failed!\n"); }*/
