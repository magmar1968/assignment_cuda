#include "hip/hip_runtime.h"
#include "../lib/support_lib/myRandom/myRandom.cuh"
#include "../lib/support_lib/myRandom/myRandom_gnr/combined.cuh"
#include "../lib/support_lib/myRandom/myRandom_gnr/tausworth.cuh"
#include "../lib/support_lib/myRandom/myRandom_gnr/linCongruential.cuh"
#include "../lib/path_gen_lib/process_eq_imp/process_eq_lognormal_multivariante.cuh"
#include "../lib/path_gen_lib/process_eq_imp/process_eq_lognormal.cuh"
#include "../lib/equity_lib/schedule_lib/schedule.cuh"
#include "../lib/equity_lib/yield_curve_lib/yield_curve.cuh"
#include "../lib/equity_lib/yield_curve_lib/yield_curve_flat.cuh"
#include "../lib/support_lib/parse_lib/parse_lib.cuh"
#include "../lib/contract_option_lib/contract_eq_option_vanilla/contract_eq_option_vanilla.cuh"
#include "../lib/support_lib/statistic_lib/statistic_lib.cuh"
#include "../lib/support_lib/timer_lib/myTimer.cuh"
//#include "../lib/support_lib/myDouble_lib/myudouble.cuh"

#define NEQ 1

struct Input_data
{
	char contract_type;
	double strike_price;
	double delta_t;
	double vol;
	char isin_code[12];
	char name[30];
	char currency[20];
	double div_yield;
	double yc;
	double start_prices[NEQ];
	double seeds[4];
};

struct Dimensions
{
	int BLOCKS;
	int TPB;
};

struct Output_data
{
	double sum;
	double sq_sum;
};

__global__ void kernel(Input_data*, Output_data*);
D void simulate_device(Input_data*, Output_data*);
H void simulate_host(Input_data*, Output_data*, Dimensions*);
HD void simulate_generic(Input_data*, Output_data*, size_t);

__global__ void kernel(Input_data* input_data, Output_data* output_data)
{
	simulate_device(input_data, output_data);
}

D void simulate_device(Input_data* input_data, Output_data* output_data)
{
	size_t index = blockIdx.x * blockDim.x + threadIdx.x;
	simulate_generic(input_data, output_data, index);
}

H void simulate_host(Input_data* input_data, Output_data* output_data, Dimensions* dimensions)
{
    for(size_t index = 0; index < dimensions->BLOCKS * dimensions->TPB; index++)
	{
		simulate_generic(input_data, output_data, index);
	}
}

HD void simulate_generic(Input_data* input_data, Output_data* output_data, size_t index)
{
	output_data[index].sum = input_data[index].strike_price;
	output_data[index].sq_sum = input_data[index].start_prices[0];
}

void Gen_dimensions(Dimensions* dim, int a, int b)
{
	dim->BLOCKS = 128 * pow(2, a);
	dim->TPB = 256 * pow(2, b);
}


int main(int argc, char** argv)
{
	hipError_t cudaStatus;
	srand(time(NULL));
	Dimensions* dim = new Dimensions;
	Timer _timer;
	for (int t = 0; t < 100; t++)
	{
		//printf("Progresso: %d di 100", t);
		
		for (size_t a = 1; a < 6; a++)
		{
			for (size_t b = 1; b < 3; b++)
			{
				Gen_dimensions(dim, a, b);
				int blocchi = dim->BLOCKS;
				int tpb = dim->TPB;

				Input_data* host_in = new Input_data[blocchi * tpb];
				Output_data* host_out = new Output_data[blocchi * tpb];

				uint seed_aus[4];
				for (size_t i = 0; i < 4; i++)
				{
					seed_aus[i] = rnd::genSeed(true);
				}
				rnd::GenCombined gnr_aus(seed_aus[0], seed_aus[1], seed_aus[2], seed_aus[3]);
				for (size_t i = 0; i < blocchi * tpb; i++)
				{
					for (size_t j = 0; j < 4; j++)
					{
						host_in[i].seeds[j] = gnr_aus.genUniformInt();
						while (host_in[i].seeds[j] <= 128)
						{
							host_in[i].seeds[j] = gnr_aus.genUniformInt();
						}
					}
					host_in[i].contract_type = 'C';
					host_in[i].strike_price = 100;
					host_in[i].delta_t = 0.2;
					host_in[i].vol = 0.0001;
					strcpy(host_in[i].isin_code, "123456789012");
					strcpy(host_in[i].name, "prova");
					strcpy(host_in[i].currency, "euro");
					host_in[i].div_yield = 0;
					host_in[i].yc = 0.05;
					host_in[i].start_prices[0] = 100;
				}

				prcr::Device dev;
				dev.CPU = false;
				dev.GPU = false;

				if (prcr::cmdOptionExists(argv, argv + argc, "-gpu"))
					dev.GPU = true;
				if (prcr::cmdOptionExists(argv, argv + argc, "-cpu"))
					dev.CPU = true;

				bool error_bool;
				error_bool = true;
				if (dev.CPU == true)
				{
					simulate_host(host_in, host_out, dim);
				}

				if (dev.GPU == true)
				{
					Input_data* dev_in;// = new Input_data[blocchi * tpb]; 
					Output_data* dev_out;//= new Output_data[blocchi * tpb];
					//Dimensions* dev_dim = new Dimensions;

					cudaStatus = hipMalloc((void**)&dev_in, blocchi * tpb * sizeof(Input_data));
					if (cudaStatus != hipSuccess) 
					{
						fprintf(stderr, "cudaMalloc1 failed!\n"); 
						printf("Errore con blocchi : %d, tpb: %d \n", blocchi, tpb);
						error_bool = false;
					}

					cudaStatus = hipMalloc((void**)&dev_out, blocchi * tpb * sizeof(Output_data));
					if (cudaStatus != hipSuccess)
					{ 
						fprintf(stderr, "cudaMalloc2 failed!\n");
						printf("Errore con blocchi : %d, tpb: %d \n", blocchi, tpb);
						error_bool = false;
					}

					/*cudaStatus = hipMalloc((void**)&dev_dim, sizeof(Dimensions));
					if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc3 failed!\n"); }*/

					cudaStatus = hipMemcpy(dev_in, host_in, blocchi * tpb * sizeof(Input_data), hipMemcpyHostToDevice);
					if (cudaStatus != hipSuccess)
					{ 
						fprintf(stderr, "cudaMemcpy1 failed! %s\n", hipGetErrorString(cudaStatus));
						printf("Errore con blocchi : %d, tpb: %d \n", blocchi, tpb);
						error_bool = false;
					}

					/*cudaStatus = hipMemcpy(dev_dim, dim, sizeof(Dimensions), hipMemcpyHostToDevice);
					if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMemcpy2 failed!\n"); }
					fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));*/

					kernel << < blocchi, tpb >> > (dev_in, dev_out);
					cudaStatus = hipGetLastError();
					if (cudaStatus != hipSuccess) 
					{
						fprintf(stderr, "Kernel failed: %s\n", hipGetErrorString(cudaStatus)); 
						printf("Errore con blocchi : %d, tpb: %d \n", blocchi, tpb);
						error_bool = false;
					}

					hipFree(dev_in);

					cudaStatus = hipMemcpy(host_out, dev_out, blocchi * tpb * sizeof(Output_data), hipMemcpyDeviceToHost);
					if (cudaStatus != hipSuccess)
					{
						fprintf(stderr, "hipMemcpy backwards failed! %s\n", hipGetErrorString(cudaStatus));
						printf("Errore con blocchi : %d, tpb: %d \n", blocchi, tpb);
						error_bool = false;
					}

					hipFree(dev_out);




				}

				for (int i = 0; i < blocchi * tpb; i++)
				{
					if (error_bool)
					{
						// std::cout << host_out[].sum <<"   "  << host_out[i].sq_sum << std::endl;
						if (host_out[i].sum - 100 != 0) { printf("errore in sum: numero blocchi %d, numero tpb %d,  indice %d\n", blocchi, tpb, i); }
						if (host_out[i].sq_sum != 100) { printf("errore in sq_sum: numero blocchi %d, numero tpb %d, indice %d\n", blocchi, tpb, i); }
					}
				}
			}
		}
	}
	delete(dim);
	_timer.Stop();
	return 0;
}
