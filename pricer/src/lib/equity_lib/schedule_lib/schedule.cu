﻿#include "schedule.cuh"

namespace pricer
{


    HD Schedule::Schedule(double t_ref, double delta_t, int dim)
        :_dim(dim)
    {
        _t = new double[_dim];
        if(delta_t<0)
        {
            for(int i = 0; i < _dim; i++)
            {
                _t[i] = 0;
                _ascending = false;
            }
        }
        else
        {
            for (int i = 0; i < _dim; i++)
            {
                _t[i] = t_ref + delta_t * i;
            }
            _ascending = true;
        }
        
        
    }


    HD Schedule::Schedule(double* t_init, int dim)
        :_dim(dim)
    {

        _t = new double[dim];
        for (int i = 0; i < _dim; i++)
        {
            _t[i] = t_init[i];
        }
        if(!Check_order())
        {
            _ascending = true;
        }
        else
        {
            
            for (int i = 0; i < _dim; i++)
            {
                _t[i] = 0;
            }
            _ascending = false;
        }
    }

    HD void Schedule::Get_t(double* ptr)
    {
        for (int i = 0; i < _dim; i++)
        {
            ptr[i] = _t[i];
        }
    }

    HD int Schedule::Get_dim(void)
    {
        return _dim;
    }

    HD bool Schedule::Check_order()
    {
        for (int i = 1; i < _dim; i++)
        {
            if (_t[i] <= _t[i - 1]) { return false; }
        }
        return true;
    }

    HD bool Get_order()
    {
        return _ascending;
    }

}