#include "hip/hip_runtime.h"
#include "option_pricer_montecarlo.cuh"

namespace prcr
{

    __host__ __device__
    Option_pricer_montecarlo::Option_pricer_montecarlo(
                    Contract_option      *contract_option,
                    Process_eq_lognormal *process,
                    size_t             N)
            :Option_pricer(contract_option,process),_N(N)
    {
        simulate_option();
    }

    __host__ __device__ double 
    Option_pricer_montecarlo::Get_price() const
    {
        return _price;
    }

    __host__ __device__ double 
    Option_pricer_montecarlo::Get_MC_error() const
    {
        return _MC_error;
    }
    
    __host__ __device__ double
    Option_pricer_montecarlo::Get_price_square() const
    {
        return _price_square;
    }

    __host__ __device__ size_t
    Option_pricer_montecarlo::Get_N() const
    {
        return _N;
    }

    __host__ __device__ void
    Option_pricer_montecarlo::Set_N(size_t N)
    {
        _N = N;
    }

    __host__ __device__ void
    Option_pricer_montecarlo::resimulate_option()
    {
        simulate_option();
    }


    __host__ __device__ void
    Option_pricer_montecarlo::simulate_option()
    {
        double * pay_off  = new double[_N];
        double * pay_off2 = new double[_N];


        Contract_eq_option &contract = 
                static_cast<Contract_eq_option&>(*_contract_option);
        Schedule * schedule = contract.Get_schedule();
        Equity_prices * starting_point = contract.Get_eq_prices();
        Path * path = new Path(starting_point,schedule, _process);

        for(size_t i = 0; i < _N; ++i)
        {

            pay_off[i] = contract.Pay_off(path);
            pay_off2[i] = pay_off[i]*pay_off[i];
        
            path->regen_path();
        }    
        
        _price = prcr::avg(pay_off,_N);
        _price_square = prcr::sum_array(pay_off2,_N);  //cos'è la somma dei quadrati ---> forse meglio cambiargli nome

        //compute_MC_error();
        delete(path);
        delete[](pay_off);delete[](pay_off2);
    }

    /**
     * @brief compute the MC error according to the formula:
     * \sigma_{MC} = \sigma/\sqrt(N)         
     * \sigma = <f^2> - <f>^2 
     */
    __host__ __device__ void
    Option_pricer_montecarlo::compute_MC_error()
    {
        _MC_error =  (_price_square/static_cast<double>(_N) - _price*_price)/ //sigma 
                                                    sqrt(static_cast<double>(_N));
    }
}



