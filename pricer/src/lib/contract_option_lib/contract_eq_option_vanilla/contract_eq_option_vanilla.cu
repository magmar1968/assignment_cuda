#include "hip/hip_runtime.h"
#include "contract_eq_option_vanilla.cuh"

namespace prcr
{

    __host__ __device__ double
    Contract_eq_option_vanilla::Pay_off(const Path *path)
    {
        double S_f = path -> Get_last_eq_price();
        return Pay_off_vanilla(S_f);
    }

    __host__ __device__ double
    Contract_eq_option_vanilla::Pay_off_vanilla(const double S_f)
    {
        

        switch (_contract_type)
        {
        case 'C':
            return Pay_off_vanilla_call(S_f);
        case 'P':
            return Pay_off_vanilla_put(S_f);
        default:
            return -1.;
        }
    }

    __host__ __device__ double
    Contract_eq_option_vanilla::Pay_off_vanilla_call(const double S_f)   
    {
        return  max(S_f - _strike_price,0.);
    }

    __host__ __device__ double 
    Contract_eq_option_vanilla::Pay_off_vanilla_put(const double S_f)
    {
        return max( _strike_price - S_f,0.);
    }
}

