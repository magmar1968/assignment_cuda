#include "hip/hip_runtime.h"
﻿#include "header.cuh"
#include <iomanip>


struct Result
{
    double p_off = 0.;
    double p_off2 =0.;
};

void __host__ print_results(std::string filename, Result *, Result *, size_t,uint);
bool __host__ run_device(prcr::Pricer_args* prcr_args, Result* host_results,uint *);
void __global__ kernel(prcr::Pricer_args* prcr_args, Result* dev_results, uint *);
bool __host__   simulate_host(prcr::Pricer_args* prcr_args, Result* host_results, uint*);
void __device__ simulate_device(prcr::Pricer_args* prcr_args, prcr::Equity_prices*, prcr::Schedule*, Result* dev_results, uint*);
void __host__ __device__ simulate_generic
(size_t, prcr::Pricer_args*, prcr::Equity_prices*, prcr::Schedule*, Result*, uint*);

__host__ bool
run_device(prcr::Pricer_args* prcr_args, Result* host_results, uint * host_seeds)
{
    using namespace prcr;
    hipError_t cudaStatus;
    Result* dev_results;
    Pricer_args* dev_prcr_args;
    uint * dev_seeds;

    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB = prcr_args->dev_opts.N_threads;

    cudaStatus = hipMalloc((void**)&dev_prcr_args, sizeof(Pricer_args));
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc1 failed!\n"); }

    cudaStatus = hipMalloc((void**)&dev_results, NBLOCKS * TPB * sizeof(Result));
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc2 failed!\n"); }

    cudaStatus = hipMalloc((void**)&dev_seeds, NBLOCKS * TPB *4 * sizeof(uint));
    if (cudaStatus != hipSuccess) { fprintf(stderr, "cudaMalloc3 failed!\n"); }




    cudaStatus = hipMemcpy(dev_prcr_args, prcr_args, sizeof(Pricer_args), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMemcpy1 failed!\n");
        fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));
    }
    cudaStatus = hipMemcpy(dev_results, host_results, NBLOCKS * TPB * sizeof(Result), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMemcpy2 failed!\n");
        fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipMemcpy(dev_seeds, host_seeds, NBLOCKS * TPB * 4 * sizeof(uint), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMemcpy3 failed!\n");
        fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));
    }



    kernel << < NBLOCKS, TPB >> > (dev_prcr_args, dev_results, dev_seeds);

    cudaStatus = hipMemcpy(host_results, dev_results, NBLOCKS * TPB * sizeof(Result), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMemcpy4 failed!\n");
        fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));
    }

    hipFree(dev_results);
    hipFree(dev_prcr_args);
    hipFree(dev_seeds);

    return cudaStatus;
}




__global__ void
kernel(prcr::Pricer_args* prcr_args, Result* dev_results, uint * dev_seeds)
{
    using namespace prcr;

    Equity_description descr(
        prcr_args->eq_descr_args.dividend_yield,
        prcr_args->eq_descr_args.rate,
        prcr_args->eq_descr_args.vol);

    Equity_prices starting_point(
        prcr_args->eq_price_args.time,
        prcr_args->eq_price_args.price,
        &descr);

    Schedule schedule(
        0.,
        prcr_args->schedule_args.T/double(prcr_args->schedule_args.dim),
        prcr_args->schedule_args.dim);

    simulate_device(prcr_args, &starting_point, &schedule, dev_results,dev_seeds);

}


__host__ bool
simulate_host(prcr::Pricer_args* prcr_args, Result* host_results, uint * host_seeds)
{
    using namespace prcr;
    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB = prcr_args->dev_opts.N_threads;

    Equity_description* descr = new Equity_description(
        prcr_args->eq_descr_args.dividend_yield,
        prcr_args->eq_descr_args.rate,
        prcr_args->eq_descr_args.vol);

    Equity_prices* starting_point = new Equity_prices(
        prcr_args->eq_price_args.time,
        prcr_args->eq_price_args.price,
        descr);

    Schedule* schedule = new Schedule(
        prcr_args->schedule_args.t_ref,
        prcr_args->schedule_args.deltat,
        prcr_args->schedule_args.dim);


    for (int index = 0; index < NBLOCKS * TPB; ++index)
    {
        simulate_generic(index, prcr_args, starting_point, schedule, host_results,host_seeds);
    }


    delete(descr);
    delete(starting_point);
    delete(schedule);
    return true; // da mettere gi� meglio
}


__device__ void
simulate_device(
    prcr::Pricer_args* prcr_args,
    prcr::Equity_prices* starting_point,
    prcr::Schedule* schedule,
    Result* dev_results,
    uint * dev_seeds)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t NBLOCKS = gridDim.x;
    size_t TPB = blockDim.x;
    if (index < NBLOCKS * TPB) simulate_generic(index, prcr_args, starting_point, schedule, dev_results,dev_seeds);
}

__host__ __device__ void
simulate_generic(size_t index,
    prcr::Pricer_args* prcr_args,
    prcr::Equity_prices* starting_point,
    prcr::Schedule* schedule,
    Result* results,
    uint * seeds)
{

    uint seed0 = seeds[0 + index * 4];
    uint seed1 = seeds[1 + index * 4];
    uint seed2 = seeds[2 + index * 4];
    uint seed3 = seeds[3 + index * 4];

    rnd::GenCombined gnr_in(seed0,seed1,seed2,seed3);


    prcr::Process_eq_lognormal process(&gnr_in, prcr_args->stc_pr_args.exact);

    prcr::Contract_eq_option_vanilla contr_opt(starting_point,
                                               schedule,
                                               prcr_args->contract_args.strike_price,
                                               prcr_args->contract_args.contract_type);
    size_t _N = prcr_args->mc_args.N_simulations;
    prcr::Option_pricer_montecarlo pricer(&contr_opt, &process, _N);

    results[index].p_off = pricer.Get_price();
    results[index].p_off2 = pricer.Get_price_square();

}




int main(int argc, char** argv)
{
    using namespace prcr;


    srand(time(NULL));


    std::string filename = "./data/infile_B3a_m_values.txt";
    std::string outfilename  = "./data/outfile_B3a_m_values.txt";
    
    Pricer_args* prcr_args = new Pricer_args;
    ReadInputOption(filename, prcr_args);

    size_t NBLOCKS = prcr_args->dev_opts.N_blocks;
    size_t TPB = prcr_args->dev_opts.N_threads;
    size_t PPT = prcr_args->mc_args.N_simulations;
    
    //gen seeds 
    srand(time(NULL));
    uint* seeds = new uint[4 * NBLOCKS * TPB];
    for (size_t inc = 0; inc < 4 * NBLOCKS * TPB; inc++)
        seeds[inc] = rnd::genSeed(true); 



    std::fstream ofs(filename.c_str(),std::fstream::out);
    ofs << "m,exact_result,exact_error,approx_result,approx_erro\n";

    for (size_t m = 0; m < 100; m+5){
        if(m == 0)
            prcr_args->schedule_args.dim = 1;
        else
            prcr_args->schedule_args.dim = m;
        

        //last_steps
        Result* exact_results = new Result[NBLOCKS * TPB];
        Result* approx_results = new Result[NBLOCKS * TPB];
        bool status = true;

        //simulate
        prcr_args->stc_pr_args.exact = true;
        status = status && run_device(prcr_args, exact_results,seeds);
        prcr_args->stc_pr_args.exact = false;
        status = status && run_device(prcr_args, exact_results,seeds);

        
        //print
        double square_sum_ex = 0., square_sum_ap = 0.;
        double final_res_ex = 0., final_res_ap = 0.;

        for(size_t i = 0; i < NBLOCKS*TPB;++i){
            final_res_ex += exact_results[i].p_off;
            final_res_ap += approx_results[i].p_off;
 
            square_sum_ex += exact_results[i].p_off2;
            square_sum_ap += approx_results[i].p_off2;
        }
        double exact_MC_error = prcr::compute_final_error(square_sum_ex,final_res_ex,NBLOCKS*TPB*PPT);
        double approx_MC_error = prcr::compute_final_error(square_sum_ap,final_res_ap,NBLOCKS*TPB*PPT);

        ofs << m
            << "," << final_res_ex/double(NBLOCKS*TPB)
            << "," << exact_MC_error
            << "," << final_res_ap/double(NBLOCKS*TPB)
            << "," << approx_MC_error << "\n";
       
        delete[](exact_results);
        delete[](approx_results);

    }
    ofs.close();

}