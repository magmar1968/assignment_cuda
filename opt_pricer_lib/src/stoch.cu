#include "hip/hip_runtime.h"
#include "../include/stoch.hpp"  //cambiare in hpp

namespace pricer
{
    HD StochProcessImp::StochProcessImp(double mu_0, double sigma_0, double S_0, double dt)
        : _mu(mu_0), _sigma(sigma_0), _S(S_0), _dt(dt)
    {
    }

    HD double StochProcessImp::getS() const
    {
         return _S;
    }

    HD double ExactSolution::get_step(const double w)
    {
       return _S = _S * exp((_mu - (_sigma*_sigma) / 2.)*_dt + _sigma*sqrt(_dt)*w);
    }


    HD EulerSolution::EulerSolution(double mu_0, double sigma_0, double S_0, double dt)
        : StochProcessImp(mu_0,sigma_0,S_0,dt)
    {
        
    }

    HD double EulerSolution::get_step(const double w)
    {
        return _S = _S * exp(_mu * _dt + _sigma * sqrt(_dt) * w );
    }



}